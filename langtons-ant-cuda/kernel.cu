#include "hip/hip_runtime.h"
﻿#include <windows.h>
#include <string>
#include <random>
#include "hip/hip_runtime.h"
#include ""

#define START_PATTERN 0 // pattern to start from
#define BATCH_SIZE 100 // patterns per batch
#define BATCH_COUNT 1 // number of batches
#define INVERT_PATTERN false // reverse and flip the pattern, for example turning "101011" into "001010"
#define MAX_ITERATIONS 100000000 // max iterations

#define GRID_SIZE (1024 & ~3) // size of the grid and resulting image rounded down to the neaest multiple of 4
#define GRID_SIZE_HALF (GRID_SIZE / 2)
#define GRID_SQUARED (GRID_SIZE * GRID_SIZE)
#define GRID_INDEX (GRID_SIZE * GRID_SIZE_HALF) + GRID_SIZE_HALF
#define FILE_SIZE (GRID_SQUARED + 310)

static const uint8_t bmp_header[54] = {
    0x42, 0x4D, // signature
    FILE_SIZE & 0xFF, (FILE_SIZE >> 8) & 0xFF, (FILE_SIZE >> 16) & 0xFF, (FILE_SIZE >> 24) & 0xFF, // file size
    0x00, 0x00, 0x00, 0x00, // reserved
    0x36, 0x01, 0x00, 0x00, // offset
    0x28, 0x00, 0x00, 0x00, // header size
    GRID_SIZE & 0xFF, (GRID_SIZE >> 8) & 0xFF, (GRID_SIZE >> 16) & 0xFF, (GRID_SIZE >> 24) & 0xFF, // width
    GRID_SIZE & 0xFF, (GRID_SIZE >> 8) & 0xFF, (GRID_SIZE >> 16) & 0xFF, (GRID_SIZE >> 24) & 0xFF, // height
    0x01, 0x00, // color planes
    0x08, 0x00, // bits per pixel
    0x00, 0x00, 0x00, 0x00, // compression
    GRID_SQUARED & 0xFF, (GRID_SQUARED >> 8) & 0xFF, (GRID_SQUARED >> 16) & 0xFF, (GRID_SQUARED >> 24) & 0xFF, // image size
    0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // ppm resolution
    0x40, 0x00, 0x00, 0x00, // number of colors
    0x00, 0x00, 0x00, 0x00 // important colors size
};

void save_bmp(const uint8_t* grid, const uint8_t* palette, const std::string& filename) {
    HANDLE handle = CreateFileA(filename.c_str(), GENERIC_WRITE, 0, NULL, CREATE_ALWAYS, FILE_ATTRIBUTE_NORMAL, NULL);
    if (handle == INVALID_HANDLE_VALUE) return;
    DWORD written;
    WriteFile(handle, bmp_header, 54, &written, NULL);
    WriteFile(handle, palette, 256, &written, NULL);
    WriteFile(handle, grid, GRID_SQUARED, &written, NULL);
    CloseHandle(handle);
}

__launch_bounds__(1)
__global__ void ant_kernel(const uint8_t* __restrict__ sizes, const uint8_t* __restrict__ patterns, uint8_t* __restrict__ grids) {
    const uint32_t i = blockIdx.x;
    const uint8_t size = sizes[i];
    const uint8_t* pattern = &patterns[i * 64];
    uint8_t* grid = &grids[(uint64_t)i * GRID_SQUARED];
    uint32_t index = GRID_INDEX;
    uint32_t ant_position_x = GRID_SIZE_HALF;
    uint32_t ant_position_y = GRID_SIZE_HALF;
    int32_t ant_direction = 1;
    uint32_t state = 0;
    for (uint64_t j = 0; j < MAX_ITERATIONS; j += 2) {
        state = grid[index];
        grid[index] = state < size ? state + 1 : 0;
        if (pattern[state]) ant_direction = -ant_direction;
        ant_position_x += ant_direction;
        if (ant_position_x >= GRID_SIZE) break;
        index = ant_position_y * GRID_SIZE + ant_position_x;
        state = grid[index];
        grid[index] = state < size ? state + 1 : 0;
        if (!pattern[state]) ant_direction = -ant_direction;
        ant_position_y += ant_direction;
        if (ant_position_y >= GRID_SIZE) break;
        index = ant_position_y * GRID_SIZE + ant_position_x;
    }
}

int main() {
    std::mt19937 gen{ std::random_device{}() };
    std::uniform_int_distribution<> dist(0, 255);
    for (uint64_t batch = 0; batch < BATCH_COUNT; ++batch) {
        const uint64_t start_pattern = START_PATTERN + (batch * BATCH_SIZE);
        const uint64_t end_pattern = start_pattern + BATCH_SIZE;
        uint64_t* valid_patterns = new uint64_t[BATCH_SIZE];
        uint64_t num_valid_patterns = 0;
        for (uint64_t i = start_pattern; i < end_pattern; ++i) {
            if (!((i + 1) & i)) continue;
            valid_patterns[num_valid_patterns++] = i;
        }
        uint8_t* sizes = new uint8_t[num_valid_patterns];
        uint8_t* patterns = new uint8_t[num_valid_patterns * 64]();
        uint8_t* palettes = new uint8_t[num_valid_patterns * 256]();
        uint8_t* grids = new uint8_t[num_valid_patterns * GRID_SQUARED]();
        for (uint64_t i = 0; i < num_valid_patterns; ++i) {
            const uint64_t pattern = valid_patterns[i];
            uint8_t size_minus_one = 0;
            for (int8_t j = 63; j >= 0; --j) {
                if (pattern & (1ULL << j)) {
                    size_minus_one = j;
                    break;
                }
            }
            const uint8_t size = size_minus_one + 1;
            sizes[i] = size - 1;
            const uint64_t i_64 = i * 64;
            for (uint8_t j = 0; j < size; ++j) {
                patterns[i_64 + (INVERT_PATTERN ? j : size_minus_one - j)] = ((pattern >> j) & 1) ^ INVERT_PATTERN;
            }
            const uint64_t i_256 = i * 256;
            const uint8_t size_4_1 = (size * 4) - 1;
            for (uint8_t j = 0; j < size_4_1; ++j) {
                palettes[i_256 + j] = dist(gen);
            }
        }
        uint8_t* d_sizes;
        uint8_t* d_patterns;
        uint8_t* d_grids;
        hipMalloc(&d_sizes, num_valid_patterns);
        hipMalloc(&d_patterns, num_valid_patterns * 64);
        hipMalloc(&d_grids, num_valid_patterns * GRID_SQUARED);
        hipMemcpy(d_sizes, sizes, num_valid_patterns, hipMemcpyHostToDevice);
        hipMemcpy(d_patterns, patterns, num_valid_patterns * 64, hipMemcpyHostToDevice);
        hipMemcpy(d_grids, grids, num_valid_patterns * GRID_SQUARED, hipMemcpyHostToDevice);
        ant_kernel << <num_valid_patterns, 1 >> > (d_sizes, d_patterns, d_grids);
        hipDeviceSynchronize();
        hipMemcpy(grids, d_grids, num_valid_patterns * GRID_SQUARED, hipMemcpyDeviceToHost);
        hipFree(d_sizes);
        hipFree(d_patterns);
        hipFree(d_grids);
        for (uint64_t i = 0; i < num_valid_patterns; ++i) {
            save_bmp(&grids[i * GRID_SQUARED], &palettes[i * 256], std::to_string(valid_patterns[i]) + ".bmp");
        }
        delete[] valid_patterns;
        delete[] sizes;
        delete[] patterns;
        delete[] palettes;
        delete[] grids;
    }
}